#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cmath>
#include <stdexcept>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <ctime>
#include <stdexcept>
#include <unordered_set>
#include <vector>
#include <random>
#include <algorithm>


#define THREADS_PER_BLOCK 256

const int clusterSize = 10000000;
const int threshold = 1000000;

struct treeNode {
    int nodeNum;
    int nodechild1;
    int nodechild2;
};

// CUDA kernel for the cluster function
__global__ void clusterKernel(int *cInstr, int numCluster, int clusterSize, int *clusterMap, int *dataset, int MAX_LEVELS, int * stopFlag, int* sharedCount ) {
    //Randomization to select 2 random 
   

    // __syncthreads(); 
    //Clustering phase
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < clusterSize) {
        if (clusterMap[idx] >= 0) {
            bool clusterFound = false;
            for (int clusterIdx = 0; clusterIdx < 3 * (numCluster); clusterIdx += 3) {
                if (cInstr[clusterIdx] == clusterMap[idx]) {
                    int distance1 = abs(dataset[cInstr[clusterIdx + 1]] - dataset[idx]);
                    int distance2 = abs(dataset[cInstr[clusterIdx + 2]] - dataset[idx]);
                    clusterMap[idx] = cInstr[clusterIdx] * 2 + (distance1 < distance2 ? 1 : 2);
                    clusterFound = true;
                    break;
                }
            }
            if (!clusterFound) {
                printf("Warning: No matching cluster found for clusterMap index %d with value %d\n", idx, clusterMap[idx]);
            }
        }
    }
    
    __syncthreads();

    // Invalidation step
    if(idx==0)
        stopFlag[0]=0;

    __syncthreads();
    

    if(idx < (1<<(MAX_LEVELS+1)))
        sharedCount[clusterMap[idx]] =0;
   
    __syncthreads();
    
    if (idx < clusterSize) 
    {
        if(clusterMap[idx]>0){
            atomicAdd(&sharedCount[clusterMap[idx]], 1);  
                if(sharedCount[clusterMap[idx]] <=threshold)
                    clusterMap[idx] = -clusterMap[idx] ;
        }
        else
            atomicAdd(&stopFlag[0], 1); 
    }


}

// Function to handle CUDA errors
void checkCudaError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("CUDA error at %s:%d: %s\n", file, line, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

#define CHECK_CUDA_ERROR(error) checkCudaError(error, __FILE__, __LINE__)

void getTwoRandomIndices(int *clusterMap, int clusterSize, int searchIndex, treeNode *node)
{

    if (!clusterMap || !node || clusterSize <= 0)
    {
        throw std::invalid_argument("Invalid arguments passed to getTwoRandomIndices");
    }

    std::unordered_set<int> uniqueIndices;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, clusterSize - 1); // Range from 0 to clusterSize - 1
    // int n = -1;
    // while (n++ < clusterSize - 1)
    // {
    //     printf("index %d value %d \n", n, clusterMap[n]);
    // }
    for (int i = 0; i < clusterSize; i++)
    {
        int indx = (dis(gen) + i) % clusterSize;
        if (clusterMap[indx] == searchIndex && uniqueIndices.find(indx) == uniqueIndices.end())
        {
            uniqueIndices.insert(indx);
            if (uniqueIndices.size() == 2)
                break;
        }
    }
    // if )
    // {
    //     throw std::runtime_error("Not enough unique indices found for the search index");
    // }
    if (uniqueIndices.size() >= 2)
    {

        auto it = uniqueIndices.begin();
        node->nodechild1 = *it++;
        node->nodechild2 = *it;
        node->nodeNum = searchIndex;
    }
}

void processClusterLevels(int *clusterMap, int clusterSize, treeNode *nodes[], int *d_dataset, int MAX_LEVELS) {
    if (!clusterMap || !nodes || !d_dataset || clusterSize <= 0) {
        throw std::invalid_argument("Invalid arguments passed to processClusterLevels");
    }

    int nodeIndex = 0;
    int *stopFlag = new int;
    int size = (1<<(MAX_LEVELS+1));
    int *sharedCount = new int[size];
    for(int i=0;i<size;i++)
        sharedCount[i]=0;

    int *d_cInstr, *d_clusterMap, *d_stopFlag,*d_sharedCount;
    for (int level = 0; level < MAX_LEVELS; level++) {
        int nodesInThisLevel = 1 << level;
        int totalInstructions = nodesInThisLevel * 3;

        int *cInstr = new int[totalInstructions];
        int instrIndex = 0;

        for (int i = 0; i < nodesInThisLevel; i++) {
            int parentIndex = (nodeIndex - 1) / 2;
            int baseClusterIndex = (level == 0) ? 0 : nodes[parentIndex]->nodeNum * 2 + i % 2 + 1;

            try {
                getTwoRandomIndices(clusterMap, clusterSize, baseClusterIndex, nodes[nodeIndex]);
            } catch (const std::exception &e) {
                printf("Error in getTwoRandomIndices: %s\n", e.what());
                delete[] cInstr;
                return;
            }

            cInstr[instrIndex++] = baseClusterIndex;
            cInstr[instrIndex++] = nodes[nodeIndex]->nodechild1;
            cInstr[instrIndex++] = nodes[nodeIndex]->nodechild2;
            nodeIndex++;
        }

        try {

            CHECK_CUDA_ERROR(hipMalloc(&d_stopFlag, sizeof(int)));
            CHECK_CUDA_ERROR(hipMemcpy(d_stopFlag, stopFlag, sizeof(int), hipMemcpyHostToDevice));

            CHECK_CUDA_ERROR(hipMalloc(&d_sharedCount, sizeof(int)));
            CHECK_CUDA_ERROR(hipMemcpy(d_clusterMapCount, sharedCount, sizeof(int), hipMemcpyHostToDevice));
            
            CHECK_CUDA_ERROR(hipMalloc(&d_cInstr, 3 * nodesInThisLevel * sizeof(int)));
            CHECK_CUDA_ERROR(hipMalloc(&d_clusterMap, clusterSize * sizeof(int)));
        
            // Copy data to device
            CHECK_CUDA_ERROR(hipMemcpy(d_cInstr, cInstr, 3 * nodesInThisLevel * sizeof(int), hipMemcpyHostToDevice));
            CHECK_CUDA_ERROR(hipMemcpy(d_clusterMap, clusterMap, clusterSize * sizeof(int), hipMemcpyHostToDevice));
        
            // Launch kernel
            int blocksPerGrid = (clusterSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            clusterKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_cInstr, nodesInThisLevel, clusterSize, d_clusterMap, d_dataset,MAX_LEVELS, d_stopFlag, d_sharedCount);
        
            CHECK_CUDA_ERROR(hipMemcpy(clusterMap, d_clusterMap, clusterSize* sizeof(int), hipMemcpyDeviceToHost));
            CHECK_CUDA_ERROR(hipMemcpy(stopFlag, d_stopFlag, sizeof(int), hipMemcpyDeviceToHost));
          
            // clusterGPU(cInstr, nodesInThisLevel, clusterSize, clusterMap, d_dataset);

            // int n = -1;
            // while (n++ < clusterSize - 1)
            // {
            //     printf("cluster value index %d value %d \n", n, clusterMap[n]);
            // }

            if(stopFlag[0] != 0)
                break;

            // Check for kernel launch errors
            CHECK_CUDA_ERROR(hipGetLastError());
        
            // Copy result back to host
          




            // clusterGPU(cInstr, nodesInThisLevel, clusterSize, clusterMap, d_dataset);
            // if (invalidateExtraOccurrences(clusterMap, clusterSize)) {
            //     delete[] cInstr;
            //     return;
            // }
        } catch (const std::exception &e) {
            printf("Error in cluster or invalidateExtraOccurrences: %s\n", e.what());
            delete[] cInstr;
            return;
        }

        delete[] cInstr;
        CHECK_CUDA_ERROR(hipFree(d_sharedCount));
        CHECK_CUDA_ERROR(hipFree(d_cInstr));
        CHECK_CUDA_ERROR(hipFree(d_stopFlag));
    }
}



int main() {
    int *boundary = new int;
    int clusterSizeVar = (int)clusterSize;
    int MAX_LEVELS = 0;
    while (clusterSizeVar >>= 1) ++MAX_LEVELS;

    std::vector<int> clustersVec(clusterSize);
    std::generate(clustersVec.begin(), clustersVec.end(), []() { return rand() % 10000 + 1; });

    int *clusters = new int[clusterSize];
    std::copy(clustersVec.begin(), clustersVec.end(), clusters);

    int *clusterMap = new int[clusterSize]();

    treeNode **nodes = new treeNode*[1 << MAX_LEVELS];
    for (int i = 0; i < 1 << MAX_LEVELS; i++) {
        nodes[i] = new treeNode();
    }

    // Allocate and copy dataset to GPU
    int *d_dataset;
    CHECK_CUDA_ERROR(hipMalloc(&d_dataset, clusterSize * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemcpy(d_dataset, clusters, clusterSize * sizeof(int), hipMemcpyHostToDevice));

    processClusterLevels(clusterMap, clusterSize, nodes, d_dataset, MAX_LEVELS);


    // int n = -1;
    // while (n++ < clusterSize - 1)
    // {
    //     printf("cluster value %d index %d value %d \n", clusters[n], n, clusterMap[n]);
    // }

    // Clean up
    for (int i = 0; i < 1 << MAX_LEVELS; i++) {
        delete nodes[i];
    }
    delete[] nodes;
    delete[] clusters;
    delete[] clusterMap;
    delete boundary;

    // Free GPU memory
    CHECK_CUDA_ERROR(hipFree(d_dataset));

    return 0;
}