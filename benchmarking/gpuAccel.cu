#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include "generateMatrix.hpp"

__global__ void matrixMultiply(const float* A, const float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            float a = A[row * n + i];
            float b = B[i * n + col];
            sum += a*b;
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int matrix_size = N * N;

    float *h_A = new float[matrix_size];
    float *h_B = new float[matrix_size]; 
    float *result = new float[matrix_size];

    generate(h_A, h_B);

    for (int iterations = 0; iterations < runs; iterations++) {
        float *d_A, *d_B, *d_C;
        hipMalloc((void**)&d_A, matrix_size * sizeof(float));
        hipMalloc((void**)&d_B, matrix_size * sizeof(float));
        hipMalloc((void**)&d_C, matrix_size * sizeof(float));

        hipMemcpy(d_A, h_A, matrix_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, matrix_size * sizeof(float), hipMemcpyHostToDevice);
        
        dim3 blockSize(16, 16);
        dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

        //for (int iterations = 0; iterations < runs; iterations++) {
            matrixMultiply<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
        //}
        
        hipDeviceSynchronize();

        hipMemcpy(result, d_C, matrix_size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
    return 0;
}
