#include "hip/hip_runtime.h"
#ifndef MASH_CUH
#include "mash.cuh"
#endif

#include <stdio.h>
#include <queue>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <chrono>
#include <iostream>

/* Note: d_aggseqLengths is the aggregated compressed length of original string (h_seqLengths)
Ex: ["dog", "mouse", "cat"] 
h_seqLengths -> [3, 5, 3] 
d_aggseqLengths -> [1, 2, 3] */
void GpuSketch::DeviceArrays::allocateDeviceArrays(uint32_t ** h_compressedSeqs, uint32_t * h_seqLengths, size_t numSequences, Param& params)
{
    hipError_t err;

    d_numSequences = numSequences;

    uint32_t kmerSize = params.kmerSize;
    size_t hashListLength = 0;   

    // Allocate memory
    err = hipMalloc(&d_aggseqLengths, numSequences*sizeof(uint32_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMalloc(&d_seqLengths, numSequences*sizeof(uint32_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    /* Flatten data */
    uint32_t * h_aggseqLengths = new uint32_t[numSequences];
    uint32_t flatStringLength=0;
    for (size_t i =0; i<numSequences; i++) flatStringLength+= (h_seqLengths[i]+15)/16;
    uint32_t * h_flattenCompressSeqs = new uint32_t[flatStringLength];
    flatStringLength=0;
    for (size_t i =0; i<numSequences; i++) 
    {
        uint32_t flatStringLengthLocal = (h_seqLengths[i]+15)/16;
        hashListLength += h_seqLengths[i] - kmerSize + 1;
        flatStringLength+=flatStringLengthLocal;
        for (size_t j=0; j<flatStringLengthLocal;j++)  
        {
            h_flattenCompressSeqs[j] = h_compressedSeqs[i][j];
            // if (i==9) printf("%u\n",h_flattenCompressSeqs[j]); 
        }
        h_flattenCompressSeqs += flatStringLengthLocal;
        h_aggseqLengths[i] = flatStringLength;
    }

    h_flattenCompressSeqs -= flatStringLength;
    //printf("%d", flatStringLength);

    err = hipMalloc(&d_hashList, hashListLength*sizeof(uint32_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }


    err = hipMalloc(&d_compressedSeqs, flatStringLength*sizeof(uint32_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_mashDist, (numSequences*(numSequences -1)/2)*sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    // Transfer data
    err = hipMemcpy(d_aggseqLengths, h_aggseqLengths, numSequences*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Gpu_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_seqLengths, h_seqLengths, numSequences*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Gpu_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_compressedSeqs, h_flattenCompressSeqs, flatStringLength*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Gpu_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    hipDeviceSynchronize();
}

void GpuSketch::DeviceArrays::deallocateDeviceArrays(){
    hipFree(d_compressedSeqs);
    hipFree(d_aggseqLengths);
    hipFree(d_seqLengths);
    hipFree(d_hashList);
    // hipFree(d_mashDist);
}

__device__ uint32_t fmix32 ( uint32_t h )
{
  h ^= h >> 16;
  h *= 0x85ebca6b;
  h ^= h >> 13;
  h *= 0xc2b2ae35;
  h ^= h >> 16;

  return h;
}

__device__ uint32_t MurmurHash3_x86_32 ( uint32_t key, int len, uint32_t seed)
{
    uint32_t h1 = seed;

    const uint32_t c1 = 0xcc9e2d51;
    const uint32_t c2 = 0x1b873593;

    uint32_t k1 = key;

    k1 *= c1;
    k1 = (k1 << 15) | (k1 >> (32 - 15));
    k1 *= c2;
    h1 ^= k1;
    h1 = (h1 << 13) | (h1 >> (32 - 13));
    h1 = h1*5+0xe6546b64;

    h1 ^= len;
    h1 = fmix32(h1);

    return h1;
} 

__global__ void sketchConstructionSerial
(
    uint32_t * d_compressedSeqs,
    uint32_t * d_aggseqLengths,
    uint32_t * d_seqLengths,
    size_t d_numSequences,
    uint32_t * d_hashList,
    uint32_t kmerSize
){
    int tx = threadIdx.x;
    int bx = blockIdx.x;


    uint32_t kmer = 0;
    uint32_t mask = (1<<2*kmerSize) - 1;

    uint32_t * hashList = d_hashList;
    uint32_t * compressedSeqs = d_compressedSeqs;

    //printf("hashList pointer in device%p\n", d_hashList);

    if (tx==0 && bx==0)
    {
        for (size_t i=0; i<d_numSequences; i++)
        {
            uint32_t seqLength = d_seqLengths[i];
            
            //if (i==9)printf("%ld:\t", i);

            for (size_t j=0; j<=seqLength-kmerSize; j++)
            {
                uint32_t index = j/16;
                uint32_t shift1 = 2*(j%16);
                if (shift1>0)
                {
                    uint32_t shift2 = 32-shift1;
                    kmer = ((compressedSeqs[index] >> shift1) | (compressedSeqs[index+1] << shift2)) & mask;
                }
                else
                {   
                    kmer = compressedSeqs[index] & mask;
                }
                uint32_t hash = MurmurHash3_x86_32  (kmer, 30, 53);
                hashList[j] = hash;
                //if (i==9) printf("(%u, %u)\t",kmer, hash);
            }
            //if (i==9) printf("\n");
            hashList += seqLength-kmerSize+1;
            compressedSeqs += (seqLength+15)/16;

        }
    }
    //printf("hashList pointer in device%p\n", d_hashList);

}


__global__ void sketchConstruction
(
    uint32_t * d_compressedSeqs,
    uint32_t * d_aggseqLengths,
    uint32_t * d_seqLengths,
    uint32_t * d_prefixHashlist,
    uint32_t * d_prefixCompressed,
    size_t d_numSequences,
    uint32_t * d_hashList,
    uint32_t kmerSize
){
    size_t tx = threadIdx.x;
    size_t bx = blockIdx.x;
    size_t threads_per_block = blockDim.x;
    size_t blocks_per_grid = gridDim.x;

    uint32_t kmer = 0;
    uint32_t mask = (1<<2*kmerSize) - 1;

    //printf("hashList pointer in device%p\n", d_hashList);

    if (bx >= d_numSequences) return;

    for (size_t j = bx; j < d_numSequences; j+=blocks_per_grid) 
    {
        uint32_t seqLength = d_seqLengths[j];
        uint32_t * hashList = d_hashList + d_prefixHashlist[j];
        uint32_t * compressedSeqs = d_compressedSeqs + d_prefixCompressed[j];
        
        for (size_t i = tx; i <= seqLength - kmerSize; i += threads_per_block) 
        {
            
            uint32_t index = i/16;
            uint32_t shift1 = 2*(i%16);
            if (shift1>0)
            {
                uint32_t shift2 = 32-shift1;
                kmer = ((compressedSeqs[index] >> shift1) | (compressedSeqs[index+1] << shift2)) & mask;
            }
            else
            {   
                kmer = compressedSeqs[index] & mask;
            }
            uint32_t hash = MurmurHash3_x86_32  (kmer, 30, 53);
            //if (j==0) printf("(%u, %u)\n",kmer, hash);
            hashList[i] = hash;
        }

    }
}


__device__ void swap(uint32_t &a, uint32_t &b) {
    uint32_t temp = a;
    a = b;
    b = temp;
}


__device__ int partition(uint32_t *arr, int low, int high) {
    uint32_t pivot = arr[high];
    int i = (low - 1);
    
    for (int j = low; j <= high- 1; j++) {
        if (arr[j] < pivot) {
            i++;
            swap(arr[i], arr[j]);
        }
    }
    swap(arr[i + 1], arr[high]);
    return (i + 1);
}



// Device function for insertion sort
__device__ void insertionSort(uint32_t *arr, int left, int right) {
    for (int i = left + 1; i <= right; i++) {
        int key = arr[i];
        int j = i - 1;
        
        while (j >= left && arr[j] > key) {
            arr[j + 1] = arr[j];
            j = j - 1;
        }
        arr[j + 1] = key;
    }
}



__device__ int findMedian(uint32_t *arr, int n) {
    insertionSort(arr, 0, n - 1);
    return arr[n / 2];
}



__device__ int quickSelect(uint32_t *arr, int left, int right, size_t k) {
    if (left == right) return arr[left];
    
    int pi = partition(arr, left, right);

    int kth = pi - left + 1;
    if (k == kth) return arr[pi];
    else if (k < kth) return quickSelect(arr, left, pi - 1, k);
    else return quickSelect(arr, pi + 1, right, k - kth);
}



__global__ void introSelectKernel(uint32_t *arr, int n, int k) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *arr = quickSelect(arr, 0, n - 1, k);
    }
}


__global__ void pruneHashes
(
    uint32_t * d_prefixHashlist,
    size_t d_numSequences,
    uint32_t * d_hashList,
    uint32_t * d_hashListPruned
){
    size_t tx = threadIdx.x;
    size_t bx = blockIdx.x;
    size_t threads_per_block = blockDim.x;
    size_t blocks_per_grid = gridDim.x;

    if (bx >= d_numSequences) return;

    for (size_t j = bx; j < d_numSequences; j+=blocks_per_grid) 
    {
        uint32_t * hashList = d_hashList + d_prefixHashlist[j];
        
        for (size_t i = tx; i < 1000; i += threads_per_block) 
        {
            d_hashListPruned[j * 1000 + i] = hashList[i];
        }

    }
}



void GpuSketch::selectNthSmallestOnGpu
(
    uint32_t* d_hashList, 
    uint32_t * h_seqLengths, 
    size_t d_numSequences, 
    Param& params
) {
    std::vector<hipStream_t> streams(d_numSequences); // tune the number of streams needed, can reuse

    uint32_t * hashList = d_hashList;
    for (size_t i = 0; i < d_numSequences; i++)
    {
        hipStreamCreate(&streams[i]);
        uint32_t numKmers = (h_seqLengths[i] - params.kmerSize + 1);
        introSelectKernel<<<params.numBlocks, params.blockSize, 0, streams[i]>>>(hashList, numKmers, 1000);
        hashList += numKmers;
    }

    // Wait for all streams to complete
    for (size_t i = 0; i < d_numSequences; ++i) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }
}




void GpuSketch::sketchConstructionOnGpu
(
    uint32_t * d_compressedSeqs,
    uint32_t * d_aggseqLengths,
    uint32_t * d_seqLengths,
    size_t d_numSequences,
    uint32_t * d_hashList,
    uint32_t * h_seqLengths,
    Param& params
){
    auto timerStart = std::chrono::high_resolution_clock::now();

    hipError_t err;

    // prefix-sum of d_seqLengths using thrust
    uint32_t * d_prefixHashlist;
    uint32_t * d_prefixCompressed;

    int bytes = d_numSequences * sizeof(uint32_t);

    hipMalloc(&d_prefixHashlist, bytes);
    hipMalloc(&d_prefixCompressed, bytes);
    

    thrust::device_ptr<uint32_t> dev_seqLengths(d_seqLengths);
    thrust::device_ptr<uint32_t> dev_prefixHash(d_prefixHashlist);
    thrust::device_ptr<uint32_t> dev_prefixComp(d_prefixCompressed);

    thrust::transform(thrust::device,
        dev_seqLengths, dev_seqLengths + d_numSequences, dev_prefixComp, 
        [] __device__ (const uint32_t& x) -> uint32_t { 
            return (x + 15) / 16;
        }
    );

    const uint32_t kmerSize = params.kmerSize; // Extract kmerSize

    thrust::transform(
        thrust::device,
        dev_seqLengths, dev_seqLengths + d_numSequences, dev_prefixHash,
        [kmerSize] __device__ (const uint32_t& x) -> uint32_t {
            return x - kmerSize + 1;
        }
    );

    thrust::exclusive_scan(dev_prefixHash, dev_prefixHash + d_numSequences, dev_prefixHash);
    thrust::exclusive_scan(dev_prefixComp, dev_prefixComp + d_numSequences, dev_prefixComp);
    auto timerEnd = std::chrono::high_resolution_clock::now();

    std::chrono::nanoseconds time = timerEnd - timerStart;
    std::cout << "Time to create prefix array: " << time.count() << "ns\n";

    

    timerStart = std::chrono::high_resolution_clock::now();
    // Serial kernel call
    //sketchConstructionSerial<<<params.numBlocks, params.blockSize>>>(d_compressedSeqs, d_aggseqLengths, d_seqLengths, d_numSequences, d_hashList, kmerSize);

    // New kernel call
    sketchConstruction<<<params.numBlocks, params.blockSize>>>(d_compressedSeqs, d_aggseqLengths, d_seqLengths, d_prefixHashlist, d_prefixCompressed, d_numSequences, d_hashList, kmerSize);
    
    hipDeviceSynchronize();

    timerEnd = std::chrono::high_resolution_clock::now();
    time = timerEnd - timerStart;
    std::cout << "Time to generate hashes: " << time.count() << "ns\n";

    hipFree(d_prefixCompressed);
    

    timerStart = std::chrono::high_resolution_clock::now();

    // // Old implementation -- for correctness
    // uint32_t * hashList = d_hashList;
    // for (size_t i = 0; i < d_numSequences; i++)
    // {
    //     thrust::device_ptr<uint32_t> hashPtr(hashList);
    //     uint32_t numKmers = (h_seqLengths[i] - kmerSize + 1);   
    //     thrust::sort(hashPtr, hashPtr + numKmers);
    //     hashList += numKmers;
    // }

    selectNthSmallestOnGpu(d_hashList, h_seqLengths, d_numSequences, params);

    timerEnd = std::chrono::high_resolution_clock::now();
    time = timerEnd - timerStart;
    std::cout << "Time to sort: " << time.count() << "ns\n";

    uint32_t * d_hashListPruned;
    bytes = d_numSequences * 1000 * sizeof(uint32_t);
    hipMalloc(&d_hashListPruned, bytes);
    uint32_t * h_pruned = (uint32_t*)malloc(bytes);
    pruneHashes<<<params.numBlocks, params.blockSize>>>(d_prefixHashlist, d_numSequences, d_hashList, d_hashListPruned);

    hipMemcpy(h_pruned, d_hashListPruned, bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < d_numSequences; i++) {
        printf("i       hashList[i] (%d)\n", i);
        for (int j = 0; j < 1000; j++) {
            printf("%d       %d\n", j, h_pruned[i * 1000 + j]);

        }
    }

    // prevent mem leak for now
    hipFree(d_hashListPruned);
    free(h_pruned);

    hipFree(d_prefixHashlist);
    

    

}

__device__ float mashDistance
(
    uint32_t * A,
    uint32_t * B,
    uint32_t kmerSize,
    uint32_t sketchSize
){
    uint32_t unionPtr = 0, APtr = 0, BPtr = 0;
    float inter=0, uni=0;

    while (true)
    {
        if ((APtr >= sketchSize) || (BPtr >= sketchSize) || (unionPtr >= sketchSize)) break;

        if (A[APtr]==B[BPtr]) 
        {   
            inter++; uni++;
            APtr++; BPtr++; unionPtr++;
        } 
        else if (A[APtr]>B[BPtr])
        {
            uni++;
            BPtr++; unionPtr++;
        }
        else
        {
            uni++;
            APtr++; unionPtr++;
        }
    }

    while (unionPtr<sketchSize-1 && (APtr<sketchSize-1 || BPtr<sketchSize-1))
    {
        if(APtr<sketchSize-1) {unionPtr++; APtr++; uni++;}
        if(BPtr<sketchSize-1) {unionPtr++; BPtr++; uni++;}
    }

    if (unionPtr<sketchSize-1)
    {
        printf("Error: Not enough hashes to build %u size union sketch\n", sketchSize);
    }
    float jaccardEstimate = (inter/uni);

    float mashDist = (log(2.0*jaccardEstimate/(1.0+jaccardEstimate)))/kmerSize;

    return mashDist;

}

__global__ void mashDistConstruction
(
    uint32_t * d_hashList,
    uint32_t * d_seqLengths,
    size_t d_numSequences,
    float * d_mashDist,
    uint32_t kmerSize,
    uint32_t sketchSize
){
    int tx = threadIdx.x;
    int bx = blockIdx.x;

    uint32_t * hashList = d_hashList;


    if (tx==0 && bx==0)
    {
        uint32_t mashDistCount=0;
        for (size_t i=0; i<d_numSequences; i++)
        {
            uint32_t * hashListStartIndex = hashList + d_seqLengths[i] - kmerSize + 1;
            for (size_t j=i+1; j<d_numSequences; j++)
            {
                float mashDist = mashDistance(hashList, hashListStartIndex, kmerSize, sketchSize);
                d_mashDist[mashDistCount++] = mashDist;
                hashListStartIndex += d_seqLengths[j] - kmerSize + 1;
            }
            hashList += d_seqLengths[i] - kmerSize + 1;
        }
    }
}


void GpuSketch::mashDistConstructionOnGpu
(   
    uint32_t * d_hashList,
    uint32_t * d_seqLengths,
    size_t d_numSequences,
    float * d_mashDist,
    uint32_t * h_seqLengths,
    Param& params
){

    mashDistConstruction<<<params.numBlocks, params.blockSize>>>(d_hashList, d_seqLengths, d_numSequences, d_mashDist, params.kmerSize, params.sketchSize);

    hipDeviceSynchronize();

}

void GpuSketch::DeviceArrays::printSketchValues(int numValues, uint32_t * h_seqLengths) 
{
    uint32_t * h_hashList = new uint32_t[numValues];

    uint32_t * hashList = d_hashList;

    hipError_t err;

    for (size_t j = 0; j < d_numSequences; j++)
    {

        err = hipMemcpy(h_hashList, hashList, numValues*sizeof(uint32_t), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr, "Gpu_ERROR: cudaMemCpy failed!\n");
            exit(1);
        }

        printf("i\thashList[i] (%zu)\n",j);
        for (int i=0; i<numValues; i++) {
            printf("%i\t%u\n", i, h_hashList[i]);
        }
        hashList += h_seqLengths[j] - 15 + 1;
       
    }

}

void GpuSketch::DeviceArrays::printMashDist(uint32_t h_numSequences) 
{
    
    float * h_mashDist = new float[h_numSequences*(h_numSequences - 1)/2];

    float * mashDist = d_mashDist;

    hipError_t err;


    err = hipMemcpy(h_mashDist, mashDist, (h_numSequences*(h_numSequences - 1)/2)*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Gpu_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    for (int i=0; i<h_numSequences; i++) 
    {
        printf("1.0\t");
        for (int j=i+1; j<h_numSequences; j++) 
        {
            printf("%f\t", *h_mashDist);
            h_mashDist++;
        }
        printf("\n");        
    }
       

}